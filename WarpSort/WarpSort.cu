#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <math.h>

#include "../utils/common.h"

#define THREADS 32
#define BLOCKS 64
#define T 64
#define K 8


int * get_splitters (int * input, int N, int s); //preliminary

__global__ void bitonic_sort_warp(int *keyin); //step1

__global__ void bitonic_warp_merge(int * keyin, int * output, int offset); //step2

__global__ void print_array_kernel(int * input, int length);

int get_length (int * array);

/******FUNCTIONS*****/

//kernel che stampa i contenuti dell'array in input
__global__ void print_array_kernel(int * input, int length){
    for(int i = 0; i < length; i++){
        printf("Array[%d] = %d \n", i, input[i]);
    }
}

//Preliminary splitter preparation function
int * get_splitters (int * input, int N, int s){
  int numElements = s * K;

  //printf("numElements = %d\n", numElements);

  size_t nBytes = numElements * sizeof(int);
	int *arrayA = (int*) malloc(nBytes);

  //seleziona a random numElements elementi da input
  for (int i = 0; i < numElements; i++){
      arrayA[i] = input[rand() % numElements];
      //printf("arrayA[%d] = %d\n", i, arrayA[i]);
  }

  // num of threads
	dim3 blocks(BLOCKS, 1);   // Number of blocks
  dim3 threads(THREADS, 1); // Number of threads

  //device memcopy
  int *bufferA, *bufferB;
  CHECK(hipMalloc((void**) &bufferA, nBytes));
  CHECK(hipMalloc((void**) &bufferB, nBytes));
	CHECK(hipMemcpy(bufferA, arrayA, nBytes, hipMemcpyHostToDevice));
  free(arrayA);

  //printf("\nbufferA prima step1 SORT\n\n");
  //print_array_kernel<<<1, 1>>>(bufferA, numElements);

  //sorting degli elementi

  bitonic_sort_warp<<<blocks, threads>>>(bufferA);

  //printf("\nbufferA dopo step1 SORT\n\n");
  //print_array_kernel<<<1, 1>>>(bufferA, numElements);

  bool isAfirst = true;
  if(numElements > 128){
    //ad ogni warp merge si inverte input ed output
    
    blocks.x = (numElements / 128) / 2;   // Number of blocks
    for(int offset = THREADS * 8; N / offset > 1; offset *= 2){
      //printf("N = %d, offset = %d, blocks.x = %d, threads.x = %d\n", N, offset, blocks, threads);
      if(isAfirst)
        bitonic_warp_merge<<<blocks, threads>>>(bufferA, bufferB, offset);
      else
        bitonic_warp_merge<<<blocks, threads>>>(bufferB, bufferA, offset);
      blocks.x = blocks.x / 2;
      
      isAfirst = !isAfirst;
    }
  }
  
  //printf("\nbufferA dopo SORT\n\n");
  //print_array_kernel<<<1, 1>>>(bufferA, numElements);

  //printf("\nbufferB dopo SORT\n\n");
  //print_array_kernel<<<1, 1>>>(bufferB, numElements);

  int * orderedSequence = (int*) malloc(numElements * sizeof(int));
  if(isAfirst){
    hipMemcpy(orderedSequence, bufferA, nBytes, hipMemcpyDeviceToHost);
  } else {
    hipMemcpy(orderedSequence, bufferB, nBytes, hipMemcpyDeviceToHost);
  }

  hipFree(bufferA);
  hipFree(bufferB);

  /*
  for (int i = 0; i < numElements; i++){
      printf("orderedSequence[%d] = %d\n", i, orderedSequence[i]);
  }*/

  //seleziona k elementi dal buffer ordinato e restituisci
  int *output = (int*) malloc(s*sizeof(int));
  int last_split = -1;
  for(int i = 0; i < s; i ++){
      output[i] = orderedSequence[i*K];
      if (output[i] == last_split){
          output[i]++;
      }
      last_split = output[i];
      printf("output[%d] = %d\n", i, output[i]);
  }

  free(orderedSequence);
   
  return output;

}

/*STEP 1:  Divide the input sequence into equal-sized subsequences. 
  Each subsequence will be sorted by an independent warp using the bitonic network.*/
__global__ void bitonic_sort_warp(int *keyin){
  //prendere thread id giusto tenendo in considerazione k_0 e k_1
  //implementare gli swap fatti bene dentro la funzione
  unsigned int id = threadIdx.x + blockDim.x * blockIdx.x;
  unsigned int subseq = id / 32; //in quale sottosequenza dell'array siamo
  unsigned int start = 128 * subseq; //primo elemento della sottosequenza da riordinare

  int i = 0, j = 0;
  int phase = 0, stage = 0;
  int k_0 = 0, k_1 = 0;
  int u = 0, index1 = 0, index2 = 0, p = 0, q = 0, m = 0, o = 0, um = 0, pm = 0;
  float dim = 0;

  //if (threadIdx.x == 0) printf("bitonic_sort_warp\n");

  //phase 0 to log(128)-1 
  for(i=2; i<128 ;i*=2){ 
    stage = 0;


    dim = i*2;
    u = ceil( (threadIdx.x+1) * (4/dim) ); //indice della sottosequenza simmetrica a cui il thread appartiene
    //printf("thread %d : u = %d \n", threadIdx.x, u);

    index1 = (u - 1) * dim;
    index2 = index1 + dim - 1;

    for(j = i/2; j > 0; j /= 2){ 
      /*
      if (threadIdx.x == 0)
        printf("thread %d : phase = %d, stage = %d \n", threadIdx.x, phase, stage);
      */
      p = threadIdx.x - (u - 1) * (dim / 4); // posizione del thread nella sottosequenza simmetrica
      //printf("thread %d : p = %d \n", threadIdx.x, p);

      //q è l'offset usato poi per k_0 e k_1
      if (stage == 0) { // primo stage della fase
          q = p;
      }
      if (stage != 0 && stage != phase){ //né primo né ultimo stage della fase
          
          //int n = 2 ^ stage; // numero di minisequenze
          m = j; // numero di freccie rosse per minisequenza
          o = j * 2; //offset speciale tra minisequenza e l'altra

          um = (int)(p / m); //indice della minisequenza a cui il thread appartiene

          pm = p - um * m; //posizione del thread nella minisequenza
          q = pm + o * um;
      }
      if (stage == phase){ //ultimo stage della fase
          q = p * 2;
      }
      k_0 = index1 + q;
      k_1 = index2 - q; 

      k_0 = start + k_0;
      k_1 = start + k_1; 
      
      //printf("thread %d : k_0 = %d, k_1 = %d \n", threadIdx.x, k_0, k_1);

      //k_0 ? position of preceding element in each pair to form ascending order
      if(keyin[k_0] > keyin[k_0+j]) {
        int tmp = keyin[k_0];
        keyin[k_0] = keyin[k_0+j];
        keyin[k_0+j] = tmp;
      }
      //k1 ? position of preceding element in each pair to form descending order
      if(keyin[k_1] > keyin[k_1-j]){
        int tmp = keyin[k_1];
        keyin[k_1] = keyin[k_1-j];
        keyin[k_1-j] = tmp;
      }

      stage++;
    }
    phase++;
  }

  stage = 0;
  //special case for the last phase 
  for(j=128/2; j>0; j/=2){
    
    dim = j * 2;
    if (dim < 4) dim = 4;
    u = ceil( (threadIdx.x+1) * (4/dim) ); //indice della sottosequenza simmetrica a cui il thread appartiene

    //printf("thread %d : u = %d \n", threadIdx.x, u);

    index1 = (u - 1) * dim;
    index2 = index1 + dim - 1;

    p = threadIdx.x - (u - 1) * (dim / 4); // posizione del thread nella sottosequenza simmetrica

    //q è l'offset usato poi per k_0 e k_1
    
    q = p;
        
    k_0 = index1 + q;
    k_1 = index2 - q; 

    k_0 = start + k_0;
    k_1 = start + k_1;

    /*
    if (threadIdx.x == 0)
        printf("thread %d : stage = %d, offset = %d \n", threadIdx.x, stage, j);
    printf("thread %d : k_0 = %d, k_1 = %d \n", threadIdx.x, k_0, k_1);
    */
      
    //k0 ? position of preceding element in the thread's first pair to form ascending order
    if(keyin[k_0] > keyin[k_0+j]){
        int tmp = keyin[k_0];
        keyin[k_0] = keyin[k_0 + j];
        keyin[k_0 + j] = tmp;
    }

    //k1 ? position of preceding element in the thread's second pair to form ascending order
    if(keyin[k_1] < keyin[k_1 - j]){
        int tmp = keyin[k_1];
        keyin[k_1] = keyin[k_1 - j];
        keyin[k_1 - j] = tmp;
    }

    stage++;
  }
}

//STEP 2: Merge all the subsequences produced in step 1 until the parallelism is insufficient.
__global__ void bitonic_warp_merge(int * keyin, int * output, int offset){
  
  int j = 0;
  int stage = 0;
  int k_0 = 0;
  int u = 0, index1 = 0, p = 0;
  float dim = 0;
  
  __shared__ int buffer[T];

  //unsigned int id = threadIdx.x + blockDim.x * blockIdx.x;
  unsigned int subseq = blockIdx.x; //in quale warp siamo
  unsigned int start = offset * subseq; //primo elemento della sottosequenza (A e B) da riordinare

  //if (threadIdx.x == 0) printf("bitonic_warp_merge, offset = %d\n", offset);

  int outIndex = start + threadIdx.x;
  int iA = start, iB = start + (offset / 2);
  int fA = start + (offset / 2), fB = start + offset;
  int tA = iA + threadIdx.x, tB = iB + threadIdx.x;
  bool compare;

  /*
  if (threadIdx.x == 0){
      printf("block %d - thread %d: subseq = %d, offset = %d\n", blockIdx.x, threadIdx.x, subseq, offset);
      printf("block %d - thread %d: iA = %d, fA = %d, iB = %d, fB = %d \n", blockIdx.x, threadIdx.x, iA, iB, fA, fB);
  }*/
    

  //printf("thread %d: tA = %d, tB = %d \n", threadIdx.x, tA, tB);

  //prendo prima sequenza di A e la prima di B e le copio sul buffer
  buffer[T/2 - 1 - threadIdx.x] = keyin[tA];
  buffer[T/2 + threadIdx.x] = keyin[tB];
  tA += THREADS;
  tB += THREADS;
  
  //A[3] < B[3]
  compare = buffer[0] < buffer[T - 1]; //se true, al prossimo caricamento prendo i primi T/2 valori di A

  int loops = 1;
  while(true)  {

    /*
    if (threadIdx.x == 0){
      printf("loop = %d,\nblock %d, thread %d, START of while: tA = %d, tB = %d \n", loops, blockIdx.x, threadIdx.x, tA, tB);
    }*/
    

    stage = 0;
    //bitonic based merge sort
    for(j = T/2; j>0; j/=2){ 
      
      dim = j * 2;
      if (dim < 2) dim = 2;
      u = ceil((threadIdx.x+1) * 2/dim); //indice della sottosequenza su cui il thread deve lavorare

      //printf("thread %d : u = %d \n", threadIdx.x, u);

      index1 = (u - 1) * dim; //primo indice della sottosequenza

      p = threadIdx.x - (u - 1) * (dim / 2); // posizione del thread nella sottosequenza simmetrica
          
      k_0 = index1 + p;

      /*
      if (threadIdx.x == 0){
        printf("block %d, thread %d : stage = %d, offset = %d \n", blockIdx.x, threadIdx.x, stage, j);
      }
      printf("block %d, thread %d : k_0 = %d \n", blockIdx.x, threadIdx.x, k_0);
      */
           
      //k0 ? position of preceding element in the thread's first pair to form ascending order
      if(buffer[k_0] > buffer[k_0 + j]){
          int tmp = buffer[k_0];
          buffer[k_0] = buffer[k_0 + j];
          buffer[k_0 + j] = tmp;
      }

      stage++;
    }
    
    //carico i primi T/2 elementi di buffer sull'output
    output[outIndex] = buffer[threadIdx.x];
    outIndex += THREADS;

    //se A e B finiscono elementi prima dell'algoritmo, prosegui solo con la sottosequenza rimanente
    if (tA > fA - 1 && tB < fB - 1)
      compare = false;
    if (tA < fA - 1 && tB > fB - 1)
      compare = true;
    if (tA > fA - 1 && tB > fB - 1){
        
      //carico gli ultimi T/2 elementi del buffer sull'output
      output[outIndex] = buffer[T/2 + threadIdx.x];
      
      break;
    }
      
    
    //usa il compare per caricare la prossima sottosequenza da A o B   
    if (compare){
      //carico T/2 elementi da A al buffer
      buffer[T/2 - 1 - threadIdx.x] = keyin[tA];
      tA += THREADS;
    } else {
      //carico T/2 elementi da B al buffer
      buffer[T/2 - 1 - threadIdx.x] = keyin[tB];
      tB += THREADS;
    }

    if (compare){ //se avevo caricato dalla sequenza A, allora Amax è il primo elemento del buffer e Bmax è l'ultimo
        compare = buffer[0] < buffer[T - 1];
    } else { //altrimenti ho caricato B sul buffer, e Amax è l'ultimo elemento, mentre Bmax è il primo
        compare = buffer[0] > buffer[T - 1];
    }

    loops++;

    /*
    if (threadIdx.x == 0){
      printf("thread %d, END of while: tA = %d, tB = %d \n",threadIdx.x, tA, tB);
    }*/
  }

}


/*******FUNZIONI DEL PROFESSORE*********/

/*The parameter dir indicates the sorting direction, ASCENDING
 or DESCENDING; if (a[i] > a[j]) agrees with the direction,
 then a[i] and a[j] are interchanged.*/
void compAndSwap(int a[], int i, int j, int dir) {
	if (dir == (a[i] > a[j])) {
		int tmp = a[i];
		a[i] = a[j];
		a[j] = tmp;
	}
}

/*It recursively sorts a bitonic sequence in ascending order,
 if dir = 1, and in descending order otherwise (means dir=0).
 The sequence to be sorted starts at index position low,
 the parameter cnt is the number of elements to be sorted.*/
void bitonicMerge(int a[], int low, int cnt, int dir) {
	if (cnt > 1) {
		int k = cnt / 2;
		for (int i = low; i < low + k; i++)
			compAndSwap(a, i, i + k, dir);
		bitonicMerge(a, low, k, dir);
		bitonicMerge(a, low + k, k, dir);
	}
}

/* This function first produces a bitonic sequence by recursively
 sorting its two halves in opposite sorting orders, and then
 calls bitonicMerge to make them in the same order */
void bitonicSort(int a[], int low, int cnt, int dir) {
	if (cnt > 1) {
		int k = cnt / 2;

		// sort in ascending order since dir here is 1
		bitonicSort(a, low, k, 1);

		// sort in descending order since dir here is 0
		bitonicSort(a, low + k, k, 0);

		// Will merge wole sequence in ascending order
		// since dir=1.
		bitonicMerge(a, low, cnt, dir);
	}
}

/*
 ******************* MAIN *****************************************************************
 */
int main(void) {
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int N = THREADS*4*BLOCKS;
	// check
	if (!(N && !(N & (N - 1)))) {
		printf("ERROR: N must be power of 2 (N = %d)\n", N);
		exit(1);
	}
	size_t nBytes = N * sizeof(int);
	int *a = (int*) malloc(nBytes);
	int *b = (int*) malloc(nBytes);

  srand ( time(NULL) );
	// fill data
	for (int i = 0; i < N; ++i) {
		//a[i] =  i%5; //rand() % 100; // / (float) RAND_MAX;
    a[i] = rand() % 100;
		b[i] = a[i];
	}

	// bitonic CPU
	double cpu_time = seconds();

  bitonicSort(b, 0, N, 1);   // 1 means sort in ascending order

	printf("CPU elapsed time: %.5f (sec)\n", seconds()-cpu_time);

	// device mem copy
	int *d_a, * d_b;
	CHECK(hipMalloc((void**) &d_a, nBytes));
  CHECK(hipMalloc((void**) &d_b, nBytes));
	CHECK(hipMemcpy(d_a, a, nBytes, hipMemcpyHostToDevice));

	// num of threads
	dim3 blocks(BLOCKS, 1);   // Number of blocks
  dim3 threads(THREADS, 1); // Number of threads
	
  /*
	int j, k;
  // external loop on comparators of size k
  for (k = 2; k <= N; k <<= 1) {
    // internal loop for comparator internal stages
    for (j = k >> 1; j > 0; j = j >> 1)
      bitonic_sort_step<<<blocks, threads * 4>>>(d_a, j, k);
  }
  */

  hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
  int l = deviceProp.multiProcessorCount; //numero di streaming multiprocessor della GPU
  for (int k = 2; k < 1000; k *= 2){ 
      if (k > l){ //TODO controllare se moltiplicando per 4 come suggerisce il paper migliora le prestazioni
          l = k;
          break;
      }
  }
  int s = l * 2; 

  printf ("\nStreaming multiprocessors = %d\n", l);
	
  
  // start computation
	hipEventRecord(start);

  /*PRELIMINARY SPLITTER STEP3*********************************************************************/
  int *output = get_splitters (a, N, s);
  
  /*STEP 1: Divide the input sequence into equal-sized subsequences. *******************************************
  Each subsequence will be sorted by an independent warp using the bitonic network.*/
  bitonic_sort_warp<<<blocks, threads>>>(d_a);

  /*STEP 2: Merge all the subsequences produced in step 1 until the parallelism is insufficient.*******************/
  //finchè il parallelismo è insufficiente, ovvero finchè N / offset >= l
  //ad ogni warp merge si inverte input ed output
  bool isAfirst = true;
  blocks.x = BLOCKS / 2;   // Number of blocks
  l = 8; //TODO rimuovere, è solo per testing!!!!!
  for(int offset = THREADS * 8; N / offset >= l ; offset *= 2){ 
    if(isAfirst)
      bitonic_warp_merge<<<blocks, threads>>>(d_a, d_b, offset);
    else
      bitonic_warp_merge<<<blocks, threads>>>(d_b, d_a, offset);
    blocks.x = blocks.x / 2;
    
    isAfirst = !isAfirst;
  }
 
  if(!isAfirst){
      int * temp = d_a;
      d_a = d_b;
      hipFree(temp);
  }
  
  /*STEP 3: Split the large subsequences produced in step 2 into small ones that can be merged independently.*******************/

  // recover data
  hipMemcpy(a, d_a, nBytes, hipMemcpyDeviceToHost);

  int s_indexes[l][s];
  int temp_i;
  for (int i = 0; i < l; i++){ //per ogni riga 
    temp_i = N / l * i;
    int splitCount = 0;
    s_indexes[i][splitCount] = temp_i; //inserisco l'indice per il primo segmento della riga
    splitCount++;
    //printf("indice %d di l = %d\n", i, s_indexes[i][0]);
    for(int j = temp_i; splitCount < s ; j++){ //calcolo gli indici dei rimanenti segmenti della riga
        if (output[splitCount] < a[j]){
            s_indexes[i][splitCount] = j;
            //printf("indice %d, %d di l, s = %d\n", i, splitCount, s_indexes[i][splitCount]);  
            splitCount++;  
        } 
    }
  }

  /****STEP 4: *************************************************************************************************/
  int *cpu_buffer; //buffer on cpu used to build the first s segment with -1 placeholders
  int *d_buffer, *d_buffer_temp;
  int s_length, global_index = 0;
  int global_s_lengths = 0;

  int *a_output;
  a_output = (int*) malloc(nBytes);

  for (int i = 0; i < s; i++){ //per ogni colonna
    //printf("\n\n---------------COLONNA---------------------- %d\n\n", i);
    cpu_buffer = (int*) malloc(l * 128 * sizeof(int));

    CHECK(hipMalloc((void**) &d_buffer, l * 128 * sizeof(int)));
    CHECK(hipMalloc((void**) &d_buffer_temp, l * 128 * sizeof(int)));

    //copia dei valori dei segmenti s in un buffer
    for (int j = 0; j < l; j++){
      if (i + 1 >= s){
        if (j + 1 >= l)
          s_length = N - s_indexes[j][i]; //caso limite ultimo segmento
        else
          s_length = s_indexes[j + 1][0] - s_indexes[j][i]; //ultimo segmento della riga
      } else{
        s_length = s_indexes[j][i + 1] - s_indexes[j][i]; //calcoliamo la lunghezza del segmento s
      }
      
      printf("segmento %d, %d: s_length = %d\n", j, i, s_length);
      if(s_length > 128){
          printf("\n\n ERRORE SEGMENTO > 128\n\n");
          break;
      }
      global_s_lengths += s_length;

      int s_index = s_indexes[j][i]; //troviamo la posizione del primo elemento del segmento s
      for (int k = 0 ; k < 128; k++){ //riempiamo il buffer con -1 e i valori del segmento s
        if (k < 128 - s_length){
          cpu_buffer[128 * j + k] = -1;
        } else {
          cpu_buffer[128 * j + k] = a[s_index];
          s_index++;
        }  
      }
    }

    
    //print 
    /*
    if (i == 0){
      printf("\n**********STAMPA DEL BUFFER PRIMA DELLO STEP2 di s(x, %d)************\n\n", i);
      for (int p = 0; p < l * 128; p++){ 
        printf("cpu_buffer[%d] = %d\n", p, cpu_buffer[p]);
      }
    }*/
    
    CHECK(hipMemcpy(d_buffer, cpu_buffer, l * 128 * sizeof(int), hipMemcpyHostToDevice));
    free(cpu_buffer);

    //fai step 2 su d_buffer (la colonna)
    blocks.x = l / 2;   // Number of blocks (warps)
    isAfirst = true;
    for(int offset = THREADS * 8; l * 128 / offset >= 1 ; offset *= 2){ 
      //printf("\nStep 2 presente!!!!\n\n" );
      if(isAfirst)
        bitonic_warp_merge<<<blocks, threads>>>(d_buffer, d_buffer_temp, offset);
      else
        bitonic_warp_merge<<<blocks, threads>>>(d_buffer_temp, d_buffer, offset);
      blocks.x = blocks.x / 2;
      
      isAfirst = !isAfirst;
    }
    
    if(!isAfirst){
      CHECK(hipMemcpy(cpu_buffer, d_buffer_temp, l * 128 * sizeof(int), hipMemcpyDeviceToHost));
    } else {
      CHECK(hipMemcpy(cpu_buffer, d_buffer, l * 128 * sizeof(int), hipMemcpyDeviceToHost));
    }
    
    /*
    //printf("\n**********STAMPA DEL BUFFER DOPO LO STEP 2 di s(x, %d)************\n\n", i);
    int num_veri = 0;
    for (int p = 0; p < l * 128; p++){
        if (cpu_buffer[p] > -1){
          //if (i == 0) printf("cpu_buffer[%d] = %d\n", p, cpu_buffer[p]);
          num_veri++;
          global_index++;
        } 
    }*/
   
    //printf("global_index nel for, colonna %d = %d\n", i, global_index);
    //printf("num_veri nel for, colonna %d = %d\n", i, num_veri);
    //printf("global_s_lengths nel for, colonna %d = %d\n", i, global_s_lengths);
    
    
    //salvo il buffer ordinato sull'output finale a rimuovendo i placeholder -1
    for(int z = 0; z < l * 128; z++){
      if (cpu_buffer[z] != -1){
        a_output[global_index] = cpu_buffer[z];
        //printf("a[%d] = %d\n", global_index, a_output[global_index]);
        global_index++;
        
      } 
    }

    hipFree(d_buffer); 
    hipFree(d_buffer_temp); 
    free(cpu_buffer);
  }

  //printf("\n\nglobal_index = %d\n", global_index);
  //printf("global_s_lengths = %d\n", global_s_lengths);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("GPU elapsed time: %.5f (sec)\n", milliseconds / 1000);

	// recover data
  //hipMemcpy(a, d_a, nBytes, hipMemcpyDeviceToHost);


	// print & check
	if (N < 100) {
		printf("GPU:\n");
		for (int i = 0; i < N; ++i){
      if(i % 128 == 0)
        printf("sottosequenza, indice = %d\n", i);
      printf("%d : %d\n", i, a[i]);
    }
			
      /*
		printf("CPU:\n");
		for (int i = 0; i < N; ++i)
			printf("%d\n", b[i]);
      */
	}
	else {
    
		for (int i = 0; i < N; ++i) {
			if (a_output[i] != b[i]) {
				printf("ERROR a[%d] != b[%d]  (a[i] = %d  -  b[i] = %d\n", i,i, a_output[i],b[i]);
				break;
			}
		}
	}

  free(a);
  free(a_output);
	hipFree(d_a);
  hipFree(d_b);
	exit(0);
}